// ICON
//
// ---------------------------------------------------------------
// Copyright (C) 2004-2024, DWD, MPI-M, DKRZ, KIT, ETH, MeteoSwiss
// Contact information: icon-model.org
//
// See AUTHORS.TXT for a list of authors
// See LICENSES/ for license information
// SPDX-License-Identifier: BSD-3-Clause
// ---------------------------------------------------------------

#include "index_list.h"

#include <hip/hip_runtime.h>
#include <cub/device/device_select.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

#include <unordered_map>
#include <memory>

// Use stream-ordered allocs if we're capturing a graph
namespace {
    bool isStreamCapturing(gpuStream_t stream) {
        hipStreamCaptureStatus captureStatus;
        hipStreamIsCapturing(stream, &captureStatus);
        return captureStatus != hipStreamCaptureStatusNone;
    }
}

class Storage {
public:
    virtual void  requestSize(size_t requestedSize) = 0;
    int* getNvalidPtr() {
        return reinterpret_cast<int*>(data);
    }
    char* getScratchPtr() {
        return data + alignment;
    }
    virtual ~Storage() = default;

protected:
    char* data = nullptr;
    static const int alignment = 512;
};

class AsyncStorage: public Storage {
public:
    AsyncStorage(gpuStream_t stream) :
      stream(stream) {    }

    void requestSize(size_t requestedSize) override final {
        if (data != nullptr) {
            hipFreeAsync(data, stream);
        }
        hipMallocAsync(&data, alignment+requestedSize, stream);
    }
    ~AsyncStorage() override {
        hipFreeAsync(data, stream);
    }

private:
    gpuStream_t stream;
};

class SyncStorage : public Storage {
public:
    void requestSize(size_t requestedSize) override final {
        if (curSize < requestedSize+alignment) {
            hipFree(data);
            hipMalloc(&data, requestedSize+alignment);
            curSize = requestedSize+alignment;
        }
    }
    ~SyncStorage() override {
        hipFree(data);
    }

private:
    size_t curSize = 0;
};

std::unordered_map<gpuStream_t, std::shared_ptr<SyncStorage>> syncStorageMap;

template<typename T>
struct ZeroCmp
{
    const T* conditions;
    const int startid;

    ZeroCmp(const int startid, const T* conditions) :
        startid(startid), conditions(conditions)
    { }

    __device__ __host__ __forceinline__
    bool operator() (const int &id)
    {
      return (conditions[ id - startid ] != 0);
    }
};

template <typename T>
static
void c_generate_index_list_gpu_generic_device(
            const T* dev_conditions,
            const int startid, const int endid,
            int* dev_indices,
            int* dev_nvalid, gpuStream_t stream)
{
    const int n = endid - startid + 1;

    // Argument is the offset of the first element
    hipcub::CountingInputIterator<int> iterator(startid);

    // Determine temporary device storage requirements
    size_t storageRequirement;
    hipcub::DeviceSelect::Flagged(nullptr, storageRequirement,
            iterator, dev_conditions, dev_indices,
            dev_nvalid, n, stream);

    // Allocate temporary storage
    // Use async storage in case we're capturing a graph
    // otherwise the sync storage per-stream
    std::shared_ptr<Storage> storage;
    if (isStreamCapturing(stream)) {
        storage = std::make_shared<AsyncStorage>(stream);
    } else {
        if (syncStorageMap.find(stream) == syncStorageMap.end()) {
            syncStorageMap[stream] = std::make_shared<SyncStorage>();
        }
        storage = syncStorageMap[stream];
    }

    storage->requestSize(storageRequirement);
    if (dev_nvalid == nullptr) {
        dev_nvalid = storage->getNvalidPtr();
    }

    ZeroCmp<T> select(startid, dev_conditions);
    hipcub::DeviceSelect::If(
            storage->getScratchPtr(), storageRequirement,
            iterator, dev_indices,
            dev_nvalid, n,
            select, stream);
}

template <typename T>
static
void c_generate_index_list_gpu_batched_generic(
            const int batch_size,
            const T* dev_conditions, const int cond_stride,
            const int startid, const int endid,
            int* dev_indices, const int idx_stride,
            int* dev_nvalid, gpuStream_t stream)
{
    for (int i = 0; i < batch_size; i++)
        c_generate_index_list_gpu_generic_device(
                dev_conditions + cond_stride*i,
                startid, endid,
                dev_indices + idx_stride*i,
                dev_nvalid + i, stream);
}

template <typename T>
static
void c_generate_index_list_gpu_generic(
            const T* dev_conditions,
            const int startid, const int endid,
            int* dev_indices, int* ptr_nvalid,
            bool copy_to_host, gpuStream_t stream)
{
    int* local_dev_nvalid = nullptr;

    c_generate_index_list_gpu_generic_device(
            dev_conditions, startid, endid, dev_indices,
            copy_to_host ? local_dev_nvalid : ptr_nvalid, stream);

    if (copy_to_host) {
        hipMemcpyAsync(ptr_nvalid, local_dev_nvalid, sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
    }
}

///
/// Exposed functions
///
/// Non-batched first
///
void c_generate_index_list_gpu_single(
            const void* dev_conditions,
            const int startid, const int endid,
            int* dev_indices, int* nvalid,
            int data_size, bool copy_to_host,
            gpuStream_t stream)
{
    switch (data_size) {
        case 1:
            c_generate_index_list_gpu_generic(
                static_cast<const char*>(dev_conditions),
                startid, endid, dev_indices, nvalid, copy_to_host, stream);
            break;
        case 4:
            c_generate_index_list_gpu_generic(
                static_cast<const int*> (dev_conditions),
                startid, endid, dev_indices, nvalid, copy_to_host, stream);
            break;
    }
}

///
/// And now batched
///
void c_generate_index_list_gpu_batched(
        const int batch_size,
        const void* dev_conditions, const int cond_stride,
        const int startid, const int endid,
        int* dev_indices, const int idx_stride,
        int* dev_nvalid, int data_size,
        gpuStream_t stream)
{
    switch (data_size) {
        case 1:
            c_generate_index_list_gpu_batched_generic(
                    batch_size,
                    static_cast<const char*>(dev_conditions),
                    cond_stride,
                    startid, endid,
                    dev_indices, idx_stride,
                    dev_nvalid, stream);
            break;

        case 4:
            c_generate_index_list_gpu_batched_generic(
                    batch_size,
                    static_cast<const int*> (dev_conditions),
                    cond_stride,
                    startid, endid,
                    dev_indices, idx_stride,
                    dev_nvalid, stream);
            break;
    }
}
